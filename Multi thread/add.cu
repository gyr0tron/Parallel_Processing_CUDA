
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  // index of current thread in block
  int index = threadIdx.x; 
  // number of threads in block
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  // 00000000 00000000 00000001 to 00010000 00000000 00000000
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays with space allocation
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  // Thread number to thread number
  add<<<1, 1024>>>(N, x, y);

  // Wait for GPU to finish before accessing on host as it doesn't block the calling CPU thread
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}